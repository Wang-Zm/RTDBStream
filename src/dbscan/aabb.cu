#include "hip/hip_runtime.h"
#include <optix.h>
#include <sutil/vec_math.h>
#include "optixScan.h"

__global__ void kGenAABB_t (
      DATA_TYPE_3* points,
      DATA_TYPE radius,
      unsigned int N,
      OptixAabb* aabb
) {
  unsigned int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if (particleIndex >= N) return;

  double3 center = {points[particleIndex].x, points[particleIndex].y, points[particleIndex].z};

  // float3 m_min = center - radius;
  // float3 m_max = center + radius;
  float3 m_min;
  float3 m_max;
  m_min.x = center.x - radius;
  m_min.y = center.y - radius;
  m_min.z = center.z - radius;
  m_max.x = center.x + radius;
  m_max.y = center.y + radius;
  m_max.z = center.z + radius;

  aabb[particleIndex] =
  {
    m_min.x, m_min.y, m_min.z,
    m_max.x, m_max.y, m_max.z
  };
}

extern "C" void kGenAABB(DATA_TYPE_3* points, DATA_TYPE width, unsigned numPrims, OptixAabb* d_aabb) {
  unsigned int threadsPerBlock = 64;
  unsigned int numOfBlocks = numPrims / threadsPerBlock + 1;

  kGenAABB_t <<<numOfBlocks, threadsPerBlock>>> (
    points,
    width,
    numPrims,
    d_aabb
    );
}

// __global__ void search_t(DIST_TYPE** dist, 
//                          DATA_TYPE** points,
//                          DATA_TYPE** queries,
//                          int query_num, 
//                          int data_num, 
//                          int bvh_num,
//                          double radius2) {
//   int left = (data_num / blockDim.x) * threadIdx.x;
//   int right = left + data_num / blockDim.x;
//   if (threadIdx.x == blockDim.x - 1) right = data_num; // 最后一个线程多一些
//   int unsigned_len = (bvh_num + 32 - 1) / 32;
//   for (int i = left; i < right; i++) {
//     for (int bvh_id = 0; bvh_id < bvh_num; bvh_id++) {
//       const DATA_TYPE point[3]    = { points[i][bvh_id * 3], 
//                                       points[i][bvh_id * 3 + 1], 
//                                       points[i][bvh_id * 3 + 2] };
//       const DATA_TYPE query[3]    = { queries[blockIdx.x][bvh_id * 3],
//                                       queries[blockIdx.x][bvh_id * 3 + 1],
//                                       queries[blockIdx.x][bvh_id * 3 + 2] };
//       const DATA_TYPE O[3]        = { query[0] - point[0], query[1] - point[1], query[2] - point[2] };
//       const DIST_TYPE sqdist      = O[0] * O[0] + O[1] * O[1] + O[2] * O[2];
//       dist[blockIdx.x][i]        += sqdist;
//       if (dist[blockIdx.x][i] >= radius2) break;
//     }
//   }
// }

// extern "C" void search_with_cuda(DIST_TYPE** dist, 
//                                  DATA_TYPE** points,
//                                  DATA_TYPE** queries, 
//                                  int query_num, 
//                                  int data_num, 
//                                  int bvh_num,
//                                  double radius2) {
//   // query, data, bvh
//   unsigned threadsPerBlock = 1024;
//   unsigned numOfBlocks = query_num;

//   // data_num / threadsPerBlock 是一个thread处理的线程数
//   search_t <<<numOfBlocks, threadsPerBlock>>> (
//     dist,
//     points,
//     queries,
//     query_num,
//     data_num,
//     bvh_num,
//     radius2
//   );
// }

__global__ void collect_t(int* label,
						  int* nn,
						  DATA_TYPE_3* window,
						  DATA_TYPE_3* out_stride,
						  int window_size,
						  int out_start,
						  int out_end,
						  DATA_TYPE_3* c_out,
						  DATA_TYPE_3* ex_cores,
						  DATA_TYPE_3* neo_cores,
						  int* c_out_num,
						  int* ex_cores_num,
						  int* neo_cores_num,
						  int min_pts) {
	int left = (window_size / blockDim.x) * threadIdx.x;
	int right = left + window_size / blockDim.x;
	if (threadIdx.x == blockDim.x - 1) right = window_size; // 最后一个线程少一些
	for (int i = left; i < right; i++) {
		if (i >= out_start && i < out_end) {
			if (label[i] == 0) {							// 原来是 core，现在 out
				int idx = atomicAdd(c_out_num, 1);
				c_out[idx] = out_stride[i - out_start]; 	// 记录 out 的部分
				idx = atomicAdd(ex_cores_num, 1);
				ex_cores[idx] = out_stride[i - out_start];
			}
			if (nn[i] > min_pts) {							// 现在是 core
				label[i] = 0;
				int idx = atomicAdd(neo_cores_num, 1);
				neo_cores[idx] = window[i];
			} else {
				label[i] = 2; 								// 现在不是 core，可暂时初始化为 noise
			}
		} else {
			if (nn[i] > min_pts && label[i] != 0) {			// 原来不是现在是
				int idx = atomicAdd(neo_cores_num, 1);
				neo_cores[idx] = window[i];
				label[i] = 0;
			} else if (nn[i] <= min_pts && label[i] == 0) { // 原来是现在不是
				int idx = atomicAdd(c_out_num, 1);
				c_out[idx] = window[i];
				label[i] = 2;								// 将 Wcurr 中 ex-core label 初始化为 noise
			}
		}
	}
}

/**
 * 1.收集 c_out, ex_cores, neo_cores
 * 2.label 设置
*/
extern "C" void set_label_collect_cores(int* label, 
										int* nn,
										DATA_TYPE_3* window,
										DATA_TYPE_3* out_stride,
										int window_size,
										int out_start,
										int out_end,
										DATA_TYPE_3* c_out,
                    DATA_TYPE_3* ex_cores,
										DATA_TYPE_3* neo_cores,
										int* c_out_num,
										int* ex_cores_num,
										int* neo_cores_num,
										int min_pts) {
  unsigned threadsPerBlock = 64;
  unsigned numOfBlocks = (window_size + threadsPerBlock - 1) / threadsPerBlock;

  collect_t <<<numOfBlocks, threadsPerBlock>>> (
	label,
	nn,	
	window,
	out_stride,
	window_size,
	out_start,
	out_end,
	c_out,
	ex_cores,
	neo_cores,
	c_out_num,
	ex_cores_num,
	neo_cores_num,
	min_pts
  );
}

__global__ void find_cores_t(int* label,
                             int* nn,
						                 int window_size,
						                 int min_pts) {
	int left = (window_size / blockDim.x) * threadIdx.x;
	int right = left + window_size / blockDim.x;
	if (threadIdx.x == blockDim.x - 1) right = window_size; // 最后一个线程少一些
	for (int i = left; i < right; i++) {
		if (nn[i] >= min_pts) {
      label[i] = 0;
    } else {
      label[i] = 2; // 初始化为 noise
    }
	}
}

extern "C" void find_cores(int* label,
                           int* nn,
                           int window_size,
                           int min_pts) {
  unsigned threadsPerBlock = 64;
  unsigned numOfBlocks = (window_size + threadsPerBlock - 1) / threadsPerBlock;
  find_cores_t <<<numOfBlocks, threadsPerBlock>>> (
    label,
    nn,
    window_size,
    min_pts
  );                        
}

__global__ void union_t(int* tmp_cluster_id, int* cluster_id, int* label, int window_size) {
	int left = (window_size / blockDim.x) * threadIdx.x;
	int right = left + window_size / blockDim.x;
	if (threadIdx.x == blockDim.x - 1) right = window_size;
	for (int i = left; i < right; i++) {
		if (label[i] == 2) {
			cluster_id[i] = -1; // noise
			continue;
		}
		int p = tmp_cluster_id[i];
		while (p != tmp_cluster_id[p]) { // 不带路径压缩的 union
			p = tmp_cluster_id[p];
		}
		cluster_id[i] = p;
	}
}

extern "C" void union_cluster(int* tmp_cluster_id, int* cluster_id, int* label, int window_size) {
  unsigned threadsPerBlock = 64;
  unsigned numOfBlocks = (window_size + threadsPerBlock - 1) / threadsPerBlock;
  union_t <<<numOfBlocks, threadsPerBlock>>> (
	tmp_cluster_id,
    cluster_id,
    label,
    window_size
  );                        
}

__global__ void find_neighbors_t(int* label, int* nn, DATA_TYPE_3* window, int window_size, DATA_TYPE radius2, int min_pts) {
	int left = (window_size / blockDim.x) * threadIdx.x;
	int right = left + window_size / blockDim.x;
	if (threadIdx.x == blockDim.x - 1) right = window_size;
	for (int i = left; i < right; i++) {
		DATA_TYPE_3 p = window[i];
		for (int j = i; i < window_size; i++) {
			DATA_TYPE_3 O = {p.x - window[j].x, p.y - window[j].y, p.z - window[j].z};
			DATA_TYPE d = O.x * O.x + O.y * O.y + O.z * O.z;
			if (d < radius2) {
				atomicAdd(nn + i, 1);
				atomicAdd(nn + j, 1);
			}
			// if (nn[i] >= min_pts) {
			// 	label[i] = 0; // core
			// 	break; // ! 不能在这停，停下之后后面的邻居无法识别到
			// }
		}
		if (nn[i] >= min_pts) {
			label[i] = 0;	// core
		} else {
			label[i] = 2;	// noise by default
		}
	}
}

extern "C" void find_neighbors(int* label, int* nn, DATA_TYPE_3* window, int window_size, DATA_TYPE radius2, int min_pts) {
	unsigned threadsPerBlock = 64;
	unsigned numOfBlocks = (window_size + threadsPerBlock - 1) / threadsPerBlock;
	find_neighbors_t <<<numOfBlocks, threadsPerBlock>>> (
		label,
		nn,
		window,
		window_size,
		radius2,
		min_pts
	);  
}

// 一开始 cluster 都是自身，这里设置自己
__global__ void set_cluster_id_t(int* label, int* cluster_id, DATA_TYPE_3* window, int window_size, DATA_TYPE radius2) {
	int left = (window_size / blockDim.x) * threadIdx.x;
	int right = left + window_size / blockDim.x;
	if (threadIdx.x == blockDim.x - 1) right = window_size;
	for (int i = left; i < right; i++) {
		DATA_TYPE_3 p = window[i];
		for (int j = i + 1; i < window_size; i++) {
			DATA_TYPE_3 O = {p.x - window[j].x, p.y - window[j].y, p.z - window[j].z};
			DATA_TYPE d = O.x * O.x + O.y * O.y + O.z * O.z;
			if (d < radius2) {
				// TODO: 可能需要原子操作
				if (i < cluster_id[j]) {
					cluster_id[j] = i;
				}
			}
		}
	}
}

extern "C" void set_cluster_id(int* label, int* cluster_id, DATA_TYPE_3* window, int window_size, DATA_TYPE radius2) {
	unsigned threadsPerBlock = 64;
	unsigned numOfBlocks = (window_size + threadsPerBlock - 1) / threadsPerBlock;
	set_cluster_id_t <<<numOfBlocks, threadsPerBlock>>> (
		label,
		cluster_id,
		window,
		window_size,
		radius2
	);
}
